
#include <hip/hip_runtime.h>
#include <__clang_cuda_builtin_vars.h>
#include <stdio.h>
#include <iostream>

struct Test{
    int a;
    int b;
};


__global__ void HelloGPU(const Test& test){
    printf("thread %d , GPU output\n", threadIdx.x);
    //std::cout << test.a <<"\n";
}

int main(void)

{
    Test test;
    
    printf("hello cuda\n");

    HelloGPU<<<1,10>>>(test);
    hipDeviceSynchronize();
    //cudaDeviceReset();
    return 0;
}
